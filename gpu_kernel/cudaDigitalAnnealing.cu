#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

texture<int, 1, hipReadModeElementType> b_text;
texture<float, 1, hipReadModeElementType> Q_text;

/**
 * @brief sum up the given aray
 *
 * @param arr input array
 * @param size the size of the array
 * @return the sum of the array
 */
float sum(float *arr, int size)
{
    float sum = 0.0f;
    for (int i = 0; i < size; i++)
    {
        sum += arr[i];
    }
    return sum;
}

/**
 * @param arr input binary array
 * @param size size of the array
 * @return the index of a random non-zero value from the array
 */
int randChoose(float *arr, int size)
{

    int nonZeroNum = 0;

    int *indicies;
    indicies = (int *)malloc(size * sizeof(int));

    for (int i = 0; i < size; i++)
    {
        if (arr[i] != 0)
        {
            indicies[nonZeroNum] = i;
            nonZeroNum++;
        }
    }

    int index = indicies[rand() % nonZeroNum];
    free(indicies);

    return index;
}

/**
 * @brief find the minimum value of the given array
 *
 * @param arr input array
 * @param size the size of the array
 * @return return the minimum value of the array
 */
float min(float *arr, int size)
{
    float min = arr[0];
    for (int i = 1; i < size; i++)
    {
        if (arr[i] < min)
        {
            min = arr[i];
        }
    }
    return min;
}

/**
 * @brief calculate the energy with given qubo matrix and binary state
 *
 * @param b array representing binary
 * @param Q qubo matrix
 * @param tempArr a temporary array to store the dot product of b^T * (Q*b)
 * @param dim dimention of the array and matrix
 */
__global__ void calculateEnergy(int *b, float *Q, float *tempArr, int dim)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim)
    {
        tempArr[i] = 0;
        for (int n = 0; n < dim; n++)
        {
            tempArr[i] += Q[i * dim + n] * b[n];
        }
        tempArr[i] = tempArr[i] * b[i];
    }
}

/**
 * @brief calculate the energy change per bit flip, record the result and return an array of the result
 *
 * @param b_copy the binary array
 * @param Q the qubo matrix
 * @param dim the dimention of the matrix and array
 * @param offset constant to deduct if the result was not accepted in the previous round
 * @param beta a factor to accept randomness
 * @param stat the array to be returned, include [0] acceptance and [1] energy change
 * @param seed a seed to create random float between (0,1] in kernel
 */
__global__ void slipBinary(int dim, float offset, float beta, float *stat, float seed)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim)
    {
        int flipped = 0;
        float delta_E;
        hiprandState state;
        hiprand_init(seed, i, 0, &state);

        // get energy change for flipping the bit [i] (check delta_E)

        // check flip
        if (tex1Dfetch(b_text, i) == 0)
        {
            flipped = 1;
        }

        for (int n = 0; n < dim; n++)
        {
            if (n == i && flipped == 1)
            {
                delta_E += tex1Dfetch(Q_text, i * dim + n); // time consuming
            }
            else
            {
                delta_E += tex1Dfetch(b_text, n) * tex1Dfetch(Q_text, i * dim + n); // time consuming
            }
        }

        if (flipped != 0)
        {
            delta_E = 2 * delta_E - offset;
        }
        else
        {
            delta_E = -2 * delta_E - offset;
        }

        // check energy or check % (check pass)
        float p = exp(-delta_E * beta);
        if (delta_E < 0)
        {
            stat[i] = 1;
        }
        else if (p > hiprand_uniform(&state))
        {
            stat[i] = 1;
        }
        else
        {
            stat[i] = 0;
        }
        stat[dim + i] = delta_E;
    }
}

/**
 * @brief create the beta array
 *
 * @param betaStart starting value of beta
 * @param betaStop ending value of beta
 * @param beta the beta array to be returned
 * @param sweeps the length of beta array
 */
void getAnnealingBeta(int betaStart, int betaStop, float *beta, int sweeps)
{

    float logBetaStart = log((float)betaStart);
    float logBetaStop = log((float)betaStop);
    float logBetaRange = (logBetaStop - logBetaStart) / (float)sweeps;
    for (int i = 0; i < sweeps; i++)
    {
        beta[i] = exp(logBetaStart + logBetaRange * i);
    }
}

/**
 * @brief the function that runs the digital annealing algorithm
 *
 * @param b binary array
 * @param Q qubo matrix
 * @param dim dimention of binary array and qubo matrix
 * @param energy energy matrix to be returned, will record energy after per flip
 * @param sweeps number of iterations to be done
 */
void digitalAnnealing(int *b, float *Q, int dim, float *energy, int sweeps)
{

    int blocks = 32 * 8;
    int threads = dim / blocks + 1;

    int betaStart = 1;
    int betaStop = 50;

    float *beta;
    beta = (float *)malloc(sweeps * sizeof(float));
    getAnnealingBeta(betaStart, betaStop, beta, sweeps);

    float offset = 0;
    float offsetIncreasingRate = 0.1;

    float *stat;
    hipMalloc(&stat, 2 * dim * sizeof(float));

    float *stat_host;
    hipHostMalloc(&stat_host, 2 * dim * sizeof(float));

    int *b_copy;
    hipMalloc(&b_copy, dim * sizeof(int));

    float *Q_copy;
    hipMalloc(&Q_copy, dim * dim * sizeof(float));
    hipMemcpy(Q_copy, Q, dim * dim * sizeof(float), hipMemcpyHostToDevice);

    // for calculating energy
    float *tempArr;
    hipMalloc(&tempArr, dim * sizeof(float));

    // for calculating energy
    float *tempArr_Host;
    hipHostMalloc(&tempArr_Host, dim * sizeof(float));

    hipBindTexture(0, b_text, b_copy);
    hipBindTexture(0, Q_text, Q_copy);

    for (int n = 0; n < sweeps; n++)
    {

        hipMemcpy(b_copy, b, dim * sizeof(int), hipMemcpyHostToDevice);

        slipBinary<<<blocks, threads>>>(dim, offset, beta[n], stat, (float)rand());
        hipDeviceSynchronize();
        hipMemcpy(stat_host, stat, 2 * dim * sizeof(float), hipMemcpyDeviceToHost);

        // stat[0] = accept, stat[1] = delta_E
        if (sum(stat_host, dim) == 0)
        {
            offset += offsetIncreasingRate * min(&stat_host[dim], dim);
        }
        else
        {
            int index = randChoose(stat_host, dim);
            b[index] = b[index] * -1 + 1;
            offset = 0;
        }

        // calculate energy ; only needed for testing
        {
            hipMemcpy(b_copy, b, dim * sizeof(int), hipMemcpyHostToDevice);
            calculateEnergy<<<blocks, threads>>>(b_copy, Q_copy, tempArr, dim);
            hipDeviceSynchronize();
            hipMemcpy(tempArr_Host, tempArr, dim * sizeof(float), hipMemcpyDeviceToHost);
            energy[n] = sum(tempArr_Host, dim);
        }
    }

    hipUnbindTexture(b_text);
    hipUnbindTexture(Q_text);

    free(beta);
    hipFree(stat);
    hipHostFree(stat_host);
    hipFree(b_copy);
    hipFree(Q_copy);
    hipFree(tempArr);
    hipHostFree(tempArr_Host);
}

/////////////////////////////////////////////////////////////////////////
/// Below is the code that Python code calls to execute the algorithm ///
/////////////////////////////////////////////////////////////////////////

extern "C"
{
    float digitalAnnealingPy(int *b, float *Q, int dim, int sweeps);
}

/**
 * @brief the function that runs the digital annealing algorithm
 *
 * @param b binary array
 * @param Q qubo matrix
 * @param dim dimention of binary array and qubo matrix
 * @param sweeps number of iterations to be done
 */
float digitalAnnealingPy(int *b, float *Q, int dim, int sweeps)
{

    srand(time(NULL));
    // srand(1);

    int blocks = 32 * 8;
    int threads = dim / blocks + 1;

    int betaStart = 1;
    int betaStop = 50;

    float *beta;
    beta = (float *)malloc(sweeps * sizeof(float));
    getAnnealingBeta(betaStart, betaStop, beta, sweeps);

    float offset = 0;
    float offsetIncreasingRate = 0.1;

    float *stat;
    hipMalloc(&stat, 2 * dim * sizeof(float));

    float *stat_host;
    hipHostMalloc(&stat_host, 2 * dim * sizeof(float));

    int *b_copy;
    hipMalloc(&b_copy, dim * sizeof(int));

    float *Q_copy;
    hipMalloc(&Q_copy, dim * dim * sizeof(float));
    hipMemcpy(Q_copy, Q, dim * dim * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(b_copy, b, dim * sizeof(int), hipMemcpyHostToDevice);

    hipBindTexture(0, b_text, b_copy);
    hipBindTexture(0, Q_text, Q_copy);

    for (int n = 0; n < sweeps; n++)
    {

        slipBinary<<<blocks, threads>>>(dim, offset, beta[n], stat, (float)rand());
        hipDeviceSynchronize();
        hipMemcpy(stat_host, stat, 2 * dim * sizeof(float), hipMemcpyDeviceToHost);

        // stat[0] = accept, stat[1] = delta_E
        if (sum(stat_host, dim) == 0)
        {
            offset += offsetIncreasingRate * min(&stat_host[dim], dim);
        }
        else
        {
            int index = randChoose(stat_host, dim);
            b[index] = b[index] * -1 + 1;
            offset = 0;
            hipMemcpy(b_copy, b, dim * sizeof(int), hipMemcpyHostToDevice);
        }
    }

    hipUnbindTexture(b_text);
    hipUnbindTexture(Q_text);

    ////////////////////////////////////////////////
    // calculate energy ; only needed for testing //
    ////////////////////////////////////////////////

    // for calculating energy
    float *tempArr;
    hipMalloc(&tempArr, dim * sizeof(float));

    // for calculating energy
    float *tempArr_Host;
    hipHostMalloc(&tempArr_Host, dim * sizeof(float));

    float energy = 0;
    {
        hipMemcpy(b_copy, b, dim * sizeof(int), hipMemcpyHostToDevice);
        calculateEnergy<<<blocks, threads>>>(b_copy, Q_copy, tempArr, dim);
        hipDeviceSynchronize();
        hipMemcpy(tempArr_Host, tempArr, dim * sizeof(float), hipMemcpyDeviceToHost);
        energy = sum(tempArr_Host, dim);
    }
    ////////////////////////////////////////////////
    ////////////////////////////////////////////////
    ////////////////////////////////////////////////

    free(beta);
    hipFree(stat);
    hipHostFree(stat_host);
    hipFree(b_copy);
    hipFree(Q_copy);
    hipFree(tempArr);
    hipHostFree(tempArr_Host);

    return energy;
}

/////////////////////////////////////////////////////////////////////////
/// Above is the code that Python code calls to execute the algorithm ///
/////////////////////////////////////////////////////////////////////////

int main()
{

    int dim = 1500;

    // create a random 1500 * 1500 array Q
    // create an inital state([1]) bit array b
    srand(1);
    float *Q;
    int *b;
    hipHostMalloc(&Q, dim * dim * sizeof(float));
    hipHostMalloc(&b, dim * sizeof(int));
    for (int i = 0; i < dim; i++)
    {
        b[i] = 1;
    }
    for (int i = 0; i < dim * dim; i++)
    {
        Q[i] = rand() / ((float)(RAND_MAX - 1) / 2 + 1) - 1;
    }

    int sweeps = 100000;
    float *energy;
    hipHostMalloc(&energy, sweeps * sizeof(float));

    digitalAnnealing(b, Q, dim, energy, sweeps);

    int stride = 1000;
    for (int i = 0; i < sweeps / stride; i++)
    {
        printf("i=%d --> e=%.5f\n", i * stride, energy[i * stride]);
    }

    hipFree(Q);
    hipFree(b);
    hipFree(energy);
    return 0;
}
