#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

texture<int, 1, hipReadModeElementType> b_text;
texture<float, 1, hipReadModeElementType> Q_text;

/**
 * @brief used to check if cuda code goes wrong
 */
void checkCudaError () {
    hipError_t err = hipGetLastError ();
    if (err != hipSuccess) {

        printf ("Cuda Error: %s, %s\n", hipGetErrorName (err), hipGetErrorString (err));
        exit (1);
    }
}

/**
 * @brief sum up the given aray
 *
 * @param arr input array
 * @param size the size of the array
 * @return the sum of the array
 */
float sum (float* arr, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        sum += arr[i];
    }
    return sum;
}

/**
 * @param arr input binary array
 * @param size size of the array
 * @return the index of a random non-zero value from the array
 */
int randChoose (float* arr, int size) {

    int nonZeroNum = 0;

    int* indicies;
    indicies = (int*)malloc (size * sizeof (int));

    for (int i = 0; i < size; i++) {
        if (abs (arr[i]) < 0.0005) {
            indicies[nonZeroNum] = i;
            nonZeroNum++;
        }
    }
    if (nonZeroNum == 0) {
        return -1;
    }
    int index = indicies[rand () % nonZeroNum];
    free (indicies);

    return index;
}

/**
 * @brief find the minimum value of the given array
 *
 * @param arr input array
 * @param size the size of the array
 * @return return the minimum value of the array
 */
float maxNum (float* arr, int size) {
    float max = arr[0];
    for (int i = 1; i < size; i++) {
        if (arr[i] > max) {
            max = arr[i];
        }
    }
    return max;
}

__global__ void dot1 (float* out, int dim) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim) {
        out[i] = 0;
        for (int j = 0; j < dim; j++) {
            out[i] += tex1Dfetch (b_text, j) * tex1Dfetch (Q_text, dim * i + j);
        }
    }
}

__global__ void dot2 (float* out, int dim) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim) {
        out[i] *= tex1Dfetch (b_text, i);
    }
}

/**
 * @brief calculate the energy change per bit flip, record the result and return an array of the result
 *
 * @param b_copy the binary array
 * @param Q the qubo matrix
 * @param dim the dimention of the matrix and array
 * @param offset constant to deduct if the result was not accepted in the previous round
 * @param beta a factor to accept randomness
 * @param stat the array to be returned, include [0] acceptance and [1] energy change
 * @param rand a random number
 */
__global__ void checkBinary (int dim, float offset, float beta, float* stat, float rand) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim - 1) {
        int flipped = 0;
        float delta_E = 0.0f;

        // check flip
        if (tex1Dfetch (b_text, i) == 0) {
            flipped = 1;
        }

        int idim = i * dim;
        for (int n = 0; n < dim; n++) {
            if (n == i && flipped == 1) {
                delta_E += tex1Dfetch (Q_text, idim + n); // time consuming
            } else {
                delta_E += tex1Dfetch (b_text, n) * tex1Dfetch (Q_text, idim + n); // time consuming
            }
        }

        if (flipped != 0) {
            delta_E = 2 * delta_E - tex1Dfetch (Q_text, idim + i) - offset;
        } else {
            delta_E = -2 * delta_E + tex1Dfetch (Q_text, idim + i) - offset;
        }

        // check energy or check % (check pass)
        stat[i] = exp (-delta_E * beta) > float (rand / float (INT_MAX)) ? 1 : 0;
        stat[dim + i] = delta_E;
    }
}

/**
 * @brief create the beta array
 *
 * @param betaStart starting value of beta
 * @param betaStop ending value of beta
 * @param beta the beta array to be returned
 * @param sweeps the length of beta array
 */
void getAnnealingBeta (float betaStart, float betaStop, float* beta, int sweeps) {

    float logBetaStart = log (betaStart);
    float logBetaStop = log (betaStop);
    float logBetaRange = (logBetaStop - logBetaStart) / (float)sweeps;
    for (int i = 0; i < sweeps; i++) {
        beta[i] = exp (logBetaStart + logBetaRange * i);
    }
}

//////////////////////////////////////////////////////////////////////////
/// Below is the code that Pythonf code calls to execute the algorithm ///
//////////////////////////////////////////////////////////////////////////

extern "C" {
    void digitalAnnealingPy (int* b, float* Q, int dim, int sweeps, float betaStart, float betaStop, int blocks, int threads);
}

/**
 * @brief the function that runs the digital annealing algorithm
 *
 * @param b binary array
 * @param Q qubo matrix
 * @param dim dimention of binary array and qubo matrix
 * @param sweeps number of iterations to be done
 */
void digitalAnnealingPy (int* b, float* Q, int dim, int sweeps, float betaStart, float betaStop, int blocks, int threads) {

    srand (time (NULL));
    // srand (1);

    float* beta;
    beta = (float*)malloc (sweeps * sizeof (float));
    getAnnealingBeta (betaStart, betaStop, beta, sweeps);

    float offset = 0;
    float offsetIncreasingRate = 0.1;

    float* stat;
    hipMalloc (&stat, 2 * dim * sizeof (float));

    float* stat_host;
    hipHostMalloc (&stat_host, 2 * dim * sizeof (float));

    int* b_copy;
    hipMalloc (&b_copy, dim * sizeof (int));
    hipMemcpy (b_copy, b, dim * sizeof (int), hipMemcpyHostToDevice);

    float* Q_copy;
    hipMalloc (&Q_copy, dim * dim * sizeof (float));
    hipMemcpy (Q_copy, Q, dim * dim * sizeof (float), hipMemcpyHostToDevice);

    hipBindTexture (0, b_text, b_copy);
    hipBindTexture (0, Q_text, Q_copy);

    // for calculating energy
    // float* tempArr;
    // hipMalloc (&tempArr, dim * sizeof (float));
    // float* tempArr_Host;
    // hipHostMalloc (&tempArr_Host, dim * sizeof (float));

    for (int n = 0; n < sweeps; n++) {
        // hipEventRecord (start1);
        checkBinary << <blocks, threads >> > (dim, offset, beta[n], stat, rand ());
        // hipEventRecord (end1);
        hipMemcpy (stat_host, stat, dim * sizeof (float), hipMemcpyDeviceToHost);

        int index = randChoose (&stat_host[dim], dim);
        if (index == -1) {
            offset += offsetIncreasingRate * maxNum (stat_host, dim);
        } else {
            b[index] = b[index] * -1 + 1;
            hipMemcpy (b_copy, b, dim * sizeof (int), hipMemcpyHostToDevice);
            offset = 0;
        }

        // if (n % 1000 == 0) {
        //     float energy = 0;
        //     dot1 << <blocks, threads >> > (tempArr, dim);
        //     hipDeviceSynchronize ();
        //     dot2 << <blocks, threads >> > (tempArr, dim);
        //     hipDeviceSynchronize ();
        //     hipMemcpy (tempArr_Host, tempArr, dim * sizeof (float), hipMemcpyDeviceToHost);
        //     energy = sum (tempArr_Host, dim);
        //     printf ("\tn = %d --> energy = %.5f\n", n, energy);
        // }

    }

    hipUnbindTexture (b_text);
    hipUnbindTexture (Q_text);

    free (beta);
    hipFree (stat);
    hipHostFree (stat_host);
    hipFree (b_copy);
    hipFree (Q_copy);
    // hipFree (tempArr);
    // hipHostFree (tempArr_Host);
}

/////////////////////////////////////////////////////////////////////////
/// Above is the code that Python code calls to execute the algorithm ///
/////////////////////////////////////////////////////////////////////////
